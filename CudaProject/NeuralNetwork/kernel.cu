#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "Matrix.h"

#include <stdio.h>

hipError_t MulMatrix(Matrix* a, Matrix* b, Matrix* &c);

__global__ void mulKernel(float *c, const float *a, const float *b, int length)
{
	int i = blockIdx.x;
	int j = blockIdx.y;

	float tmp = 0;

	for(int l=0;l<length;l++)
	{
		tmp += a[i*length+l]*b[l*length+j];
	}

	c[512*i+j] = tmp;
}

int main()
{
    const int arraySize = 512;
    Matrix* matrixA = new Matrix(arraySize,arraySize);
	Matrix* matrixB = new Matrix(arraySize,arraySize);
	Matrix* matrixC;

	for (int i=0;i<arraySize;i++)
	{
		for (int j=0;j<arraySize;j++)
		{
			matrixA->setValue(i,j,1.1);
			matrixB->setValue(i,j,1.1);
		}
	}

    // Add vectors in parallel.
    hipError_t cudaStatus = MulMatrix(matrixA,matrixB,matrixC);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


	delete matrixA;
	delete matrixB;
	delete matrixC;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t MulMatrix(Matrix* a, Matrix* b, Matrix* &c)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, a->getDimY() * b->getDimX() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_a, a->getDimX() * a->getDimY() * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_b, b->getDimX() * b->getDimY() * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a->getArrayPointer(), a->getDimX() * a->getDimY() * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_b, b->getArrayPointer(), b->getDimX() * b->getDimY() * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dim3 gridSize(b->getDimX(),a->getDimY(),1);

    // Launch a kernel on the GPU with one thread for each element.
    mulKernel<<<gridSize, 1>>>(dev_c, dev_a, dev_b, a->getDimX());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	c = new Matrix(b->getDimX(),a->getDimY());
	cudaStatus = hipMemcpy(c->getArrayPointer(), dev_c, a->getDimY() * b->getDimX() * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
